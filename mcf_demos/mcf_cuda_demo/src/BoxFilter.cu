#include "hip/hip_runtime.h"
/**
 * See header file for documentation.
 *
 * Copyright (c) 2024 Accenture
 */
#include "mcf_cuda_demo/BoxFilter.h"
#include "mcf_cuda/CudaErrorHelper.h"

#include <stdio.h>


namespace {

struct ImageSize
{
    ImageSize(uint16_t widthIn, uint16_t heightIn, uint8_t numChannelsIn)
    : width(widthIn)
    , height(heightIn)
    , numChannels(numChannelsIn)
    , size(widthIn * numChannelsIn * heightIn) 
    , pitch(widthIn * numChannelsIn) {};

    const uint16_t width;
    const uint16_t height;
    const uint8_t numChannels;
    const uint32_t size;
    const uint32_t pitch;
};


__global__ void blurRegion(
    const uint8_t* const rawImgBuffer,
    uint8_t* const filteredImgBuffer,
    const ImageSize inputImageSize,
    const ImageSize outputImageSize,
    const int kernelSize)
{
    const int outputIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (outputIdx >= outputImageSize.size)
    {
        return;
    }

    const uint32_t rowIdx = outputIdx / outputImageSize.pitch;
    const uint32_t colIdx = outputIdx % outputImageSize.pitch;

    const int halfSize = kernelSize / 2;

    const size_t inputImgRowIdx = rowIdx + halfSize;
    const size_t inputImgColIdx = colIdx + (halfSize * inputImageSize.numChannels);

    // Get the average of the pixels in the kernel neighbourhood in the original image.
    int pixelSum = 0;
    for (int i=-halfSize; i<halfSize + 1; ++i)
    {
        for (int j=-halfSize; j<halfSize + 1; ++j)
        {
            const int currentRowIdx = inputImgRowIdx + j;
            const int currentColIdx = inputImgColIdx + (i * outputImageSize.numChannels);
            const int inputIdx = currentRowIdx * inputImageSize.pitch + currentColIdx;
            pixelSum += rawImgBuffer[inputIdx];
        }
    }
    filteredImgBuffer[outputIdx] = __float2uint_rn((float) pixelSum / (kernelSize * kernelSize));
}

} // namespace


namespace mcf_cuda_demo {

BoxFilter::BoxFilter(const int cudaDeviceId) : fCudaDeviceId(cudaDeviceId) {}


BoxFilter::~BoxFilter() = default;


values::mcf_cuda_demo_value_types::demo_types::DemoImageUint8 BoxFilter::blurImage(
    const DemoImageUint8& image,
    const int kernelSize)
{
    MCF_CHECK_CUDA(hipSetDevice(fCudaDeviceId));
    hipDeviceProp_t deviceProp;
    MCF_CHECK_CUDA(hipGetDeviceProperties(&deviceProp, fCudaDeviceId));
    MCF_CHECK_CUDA_ERROR;

    const int numChannels = (image.format == values::mcf_cuda_demo_value_types::demo_types::DemoImgFormat::GRAY) ? 1 : 3;
    
    const ImageSize inputImageSize(image.width, image.height, numChannels);
    fRawImage.increase(inputImageSize.size);

    const ImageSize outputImageSize(
        inputImageSize.width - kernelSize + 1,
        inputImageSize.height - kernelSize + 1,
        numChannels);

    // Copy image into pre-allocated GPU memory
    MCF_CHECK_CUDA(hipMemcpy(
        fRawImage.get(), 
        image.extMemPtr(), 
        inputImageSize.size, 
        hipMemcpyHostToDevice));
    MCF_CHECK_CUDA_ERROR;

    int blockSize = deviceProp.maxThreadsPerBlock;
    int numBlocks = (outputImageSize.size + blockSize - 1) / blockSize;

    // Allocate memory for output image in GPU memory
    mcf::cuda::unique_array<uint8_t> hipArray = mcf::cuda::make_array<uint8_t>(outputImageSize.size);

    // Launch cuda kernel to blur image
    blurRegion<<<numBlocks, blockSize>>>(
        fRawImage.get(),
        hipArray.get(),
        inputImageSize,
        outputImageSize,
        kernelSize
    );
    MCF_CHECK_CUDA_ERROR;

    // Initialise the output MCF value.
    DemoImageUint8 blurredImage(
        outputImageSize.width, 
        outputImageSize.height, 
        outputImageSize.pitch, 
        image.format, 
        image.timestamp);

    // Initialise ExtMemValue of the output MCF value with the blurred image buffer in GPU memory.
    blurredImage.extMemInit(std::move(hipArray));
    MCF_CHECK_CUDA_ERROR;

    return blurredImage;
}

} // namespace mcf_cuda_demo
